#include "hip/hip_runtime.h"
#include <array>
#include "hip/hip_vector_types.h"
#include "hip/hip_runtime.h"

#include <fstream>

typedef std::chrono::high_resolution_clock Clock;

void initialize(REAL* positions, int N) {
  const REAL space = 10;
  int i = 0;
  for (size_t x = 0; x < N; x++) {
    REAL x_pos = x * space;
    for (size_t y = 0; y < N; y++) {
      REAL y_pos = y * space;
      for (size_t z = 0; z < N; z++) {
        positions[3*i+0] = x_pos;
        positions[3*i+1] = y_pos;
        positions[3*i+2] = z * space;
        i++;
      }
    }
  }
}

REAL compute_sum_cuda(REAL* voa, int N) {
  REAL sum = 0.0;

  for (int j = 0; j < N*N*N; j++) {
    sum += fabs(voa[3*j+0]);
    sum += fabs(voa[3*j+1]);
    sum += fabs(voa[3*j+2]);
  }
  return sum;
}

void clear_force(REAL* voa, int N) {
  for (int j = 0; j < N*N*N; j++) {
    voa[3*j+0] = 0;
    voa[3*j+1] = 0;
    voa[3*j+2] = 0;
  }
}

bool are_same(REAL a, REAL b) {
  return fabs(a - b) < std::numeric_limits<REAL>::epsilon();
}

__device__ int3 get_box_coordinates(REAL3 pos, int32_t* grid_dimensions, uint32_t box_length) {
  int3 box_coords;
  box_coords.x = (floor(pos.x) - grid_dimensions[0]) / box_length;
  box_coords.y = (floor(pos.y) - grid_dimensions[1]) / box_length;
  box_coords.z = (floor(pos.z) - grid_dimensions[2]) / box_length;
  return box_coords;
}

__device__ uint32_t get_box_id_2(int3 bc, uint32_t* num_boxes_axis) {
  return bc.z * num_boxes_axis[0]*num_boxes_axis[1] + bc.y * num_boxes_axis[0] + bc.x;
}

__device__ uint32_t get_box_id(REAL3 pos, uint32_t* num_boxes_axis, int32_t* grid_dimensions, uint32_t box_length) {
  int3 box_coords = get_box_coordinates(pos, grid_dimensions, box_length);
  return get_box_id_2(box_coords, num_boxes_axis);
}

__device__ void compute_force(REAL* positions, REAL* diameters, uint32_t idx, uint32_t nidx, REAL* result) {
  REAL r1 = 0.5 * diameters[idx];
  REAL r2 = 0.5 * diameters[nidx];
  // We take virtual bigger radii to have a distant interaction, to get a desired density.
  REAL additional_radius = 10.0 * 0.15;
  r1 += additional_radius;
  r2 += additional_radius;

  REAL comp1 = positions[3*idx + 0] - positions[3*nidx + 0];
  REAL comp2 = positions[3*idx + 1] - positions[3*nidx + 1];
  REAL comp3 = positions[3*idx + 2] - positions[3*nidx + 2];
  REAL center_distance = sqrtf(comp1 * comp1 + comp2 * comp2 + comp3 * comp3);

  // the overlap distance (how much one penetrates in the other)
  REAL delta = r1 + r2 - center_distance;

  if (delta < 0) {
    return;
  }

  // to avoid a division by 0 if the centers are (almost) at the same location
  if (center_distance < 0.00000001) {
    result[3*idx + 0] += 42.0;
    result[3*idx + 1] += 42.0;
    result[3*idx + 2] += 42.0;
    return;
  }

  // printf("Colliding cell [%d] and [%d]\n", idx, nidx);
  // printf("Delta for neighbor [%d] = %f\n", nidx, delta);

  // the force itself
  REAL r = (r1 * r2) / (r1 + r2);
  REAL gamma = 1; // attraction coeff
  REAL k = 2;     // repulsion coeff
  REAL f = k * delta - gamma * sqrt(r * delta);

  REAL module = f / center_distance;
  result[3*idx + 0] += module * comp1;
  result[3*idx + 1] += module * comp2;
  result[3*idx + 2] += module * comp3;
}


__device__ void default_force(REAL* positions,
                   REAL* diameters,
                   uint32_t idx, uint32_t start, uint16_t length,
                   uint32_t* successors,
                   REAL* result) {
  // printf("start = %d \n", start);
  // printf("length = %d \n", length);
  uint32_t nidx = start;
  for (uint16_t nb = 0; nb < length; nb++) {
    // implement logic for within radius here
    if (nidx != idx) {
      compute_force(positions, diameters, idx, nidx, result);
    }
    // traverse linked-list
    nidx = successors[nidx];
  }
  // printf("\n");
}

__global__ void collide(
       REAL* positions,
       REAL* diameters,
       int N,
       uint32_t* starts,
       uint16_t* lengths,
       uint32_t* successors,
       uint32_t* box_length,
       uint32_t* num_boxes_axis,
       int32_t* grid_dimensions,
       REAL* result) {
  uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tidx < N * N * N) {
    // if (tidx == 0) {
      REAL3 pos;
      pos.x = positions[3*tidx + 0];
      pos.y = positions[3*tidx + 1];
      pos.z = positions[3*tidx + 2];

      int3 box_coords = get_box_coordinates(pos, grid_dimensions, box_length[0]);

      // Moore neighborhood
      for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
          for (int x = -1; x <= 1; x++) {
            uint32_t bidx = get_box_id_2(box_coords + make_int3(x, y, z), num_boxes_axis);
            if (lengths[bidx] != 0) {
              // printf("Box %d\n", bidx);
              // printf("length = %d\n", lengths[bidx]);
              default_force(positions, diameters, tidx, starts[bidx], lengths[bidx], successors, result);
            }
          }
        }
      }
    }
  // }
}

int cuda_collide(std::vector<uint32_t>* starts,
                 std::vector<uint16_t>* lengths,
                 std::vector<uint32_t>* successors,
                 uint32_t box_length,
                 std::array<uint32_t, 3>* num_boxes_axis,
                 std::array<int32_t, 3>* grid_dimensions,
                 int N, int T, int diameter, REAL expected) {
  REAL* positions;
  REAL* force;
  REAL* diameters;

  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&positions, 3*N*N*N*sizeof(REAL));
  hipMallocManaged(&force, 3*N*N*N*sizeof(REAL));
  hipMallocManaged(&diameters, N*N*N*sizeof(REAL));

  uint32_t* d_starts = NULL;
  uint16_t* d_lengths = NULL;
  uint32_t* d_sucessors = NULL;
  uint32_t* d_box_length = NULL;
  uint32_t* d_num_boxes_axis = NULL;
  int32_t* d_grid_dimensions = NULL;

  hipMalloc(&d_starts, starts->size() * sizeof(uint32_t));
  hipMalloc(&d_lengths, lengths->size() * sizeof(uint16_t));
  hipMalloc(&d_sucessors, successors->size() * sizeof(uint32_t));
  hipMalloc(&d_box_length, sizeof(uint32_t));
  hipMalloc(&d_num_boxes_axis, 3 * sizeof(uint32_t));
  hipMalloc(&d_grid_dimensions, 3 * sizeof(int32_t));

  hipMemcpy(d_starts, starts->data(), starts->size() * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_lengths, lengths->data(), lengths->size() * sizeof(uint16_t), hipMemcpyHostToDevice);
  hipMemcpy(d_sucessors, successors->data(), successors->size() * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_box_length, &box_length, sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_num_boxes_axis, num_boxes_axis->data(), 3 * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_grid_dimensions, grid_dimensions->data(), 3 * sizeof(uint32_t), hipMemcpyHostToDevice);

  // auto total_mem = 3*N*N*N*sizeof(REAL) + 3*N*N*N*sizeof(REAL) + N*N*N*sizeof(REAL);
  // std::cout << "total memory allocated = " << total_mem / (1024*1024) << " MB" << std::endl;
 
  // initialize
  const REAL space = 20;
  int i = 0;
  for (size_t x = 0; x < N; x++) {
    REAL x_pos = x * space;
    for (size_t y = 0; y < N; y++) {
      REAL y_pos = y * space;
      for (size_t z = 0; z < N; z++) {
        positions[3*i+0] = x_pos;
        positions[3*i+1] = y_pos;
        positions[3*i+2] = z * space;
        i++;
      }
    }
  }

  for (int j = 0; j < N*N*N; j++) {
    diameters[j] = diameter;
  }

  for (int t = 0; t < T; t++) {
    auto t1 = Clock::now();
    
    // Launch kernel
    int blockSize = 1024;
    int numBlocks = (N*N*N + blockSize - 1) / blockSize;
    collide<<<numBlocks, blockSize>>>(positions, diameters, N, d_starts, d_lengths, d_sucessors, d_box_length, d_num_boxes_axis, d_grid_dimensions, force);
   
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    auto t2 = Clock::now();
    std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() << " ms" << std::endl;

    // remove("cuda.txt");
    // std::ofstream ofs("cuda.txt", std::ofstream::out);
    // for (int k = 0; k < N*N*N; k++) {
    //   ofs << force[3*k + 0] << ", " << force[3*k + 1] << ", " << force[3*k + 2] << std::endl;
    // }
    // ofs.close();

    REAL actual = compute_sum_cuda(force, N);
    if (are_same(actual, expected)) {
      std::cout << "Correct result! Because " << std::setprecision(15) << actual << " == " << expected << std::endl;
      clear_force(force, N);
      continue;    
    } else {
      std::cout << "Got result = " << actual << std::endl;
      std::cout << "Wrong result! Difference = " << fabs(actual - expected) << std::endl;
      std::cout << "Should be " << expected << std::endl;
      return 1;
    }
  }
 
  // Free memory
  hipFree(positions);
  hipFree(force);
  hipFree(diameters);
  hipFree(d_starts);
  hipFree(d_lengths);
  hipFree(d_box_length);
  hipFree(d_sucessors);
  hipFree(d_grid_dimensions);
  hipFree(d_num_boxes_axis);

  return 0;
}
