#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

typedef std::chrono::high_resolution_clock Clock;

void initialize(float* positions, int N) {
  const float space = 10;
  int i = 0;
  for (size_t x = 0; x < N; x++) {
    float x_pos = x * space;
    for (size_t y = 0; y < N; y++) {
      float y_pos = y * space;
      for (size_t z = 0; z < N; z++) {
        positions[3*i+0] = x_pos;
        positions[3*i+1] = y_pos;
        positions[3*i+2] = z * space;
        i++;
      }
    }
  }
}

float compute_sum(float* voa, int N) {
  float sum = 0.0;

  for (int j = 0; j < N*N*N; j++) {
    sum += voa[3*j+0];
    sum += voa[3*j+1];
    sum += voa[3*j+2];
  }
  return sum;
}

// make list of randomly ordered indices (rai)
void make_rai(int* rai, int N, int cpc) {
  std::random_device rd;
  std::mt19937 rng(time(0));
  std::uniform_int_distribution<int> uni(0, N * N * N);

  for (int i = 0; i < cpc * N * N * N; i++) {
    rai[i] = uni(rng);
  }
}

// make list of regular ordered indices (rei)
void make_rei(int* rei, int N, int cpc) {
  for (int i = 0; i < N * N * N; i++) {
    if (i < cpc / 2) {
      for (int k = 0; k < cpc; k++) {
        rei[i * cpc + k] = k;
      }
    }
    else if (i >= N*N*N - cpc / 2) {
      for (int k = 0; k < cpc; k++) {
        rei[i * cpc + k] = N*N*N - cpc + k;
      }
    } else {
      int idx = 0;
      for (int j = -cpc / 2; j < cpc / 2 + 1; j++) {
        if (j != 0) {
          if (((i + j) >= 0) && ((i + j) < N*N*N)) {
            rei[i * cpc + idx] = i + j;
          } else if (((i + j) == N*N*N)) {
            rei[i * cpc + idx] = N*N*N - 1;
          } else {
            rei[i * cpc + idx] = i * cpc + idx;
          }
          idx++;
        }
      }
    }
  }
}

__global__ void collide(
       float* positions,
       float* diameters,
       float* result,
       int* nidc,
       int N,
       int cpc) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N*N*N) {
    for (int nb = 0; nb < cpc; nb++) {
      float r1 = 0.5 * diameters[i];
      float r2 = 0.5 * diameters[nidc[cpc * i + nb]];
      // We take virtual bigger radii to have a distant interaction, to get a
      // desired density.
      float additional_radius = 10.0 * 0.15;
      r1 += additional_radius;
      r2 += additional_radius;

      float comp1 = positions[3*i + 0] - positions[3*nidc[cpc * i + nb]+0];
      float comp2 = positions[3*i + 1] - positions[3*nidc[cpc * i + nb]+1];
      float comp3 = positions[3*i + 2] - positions[3*nidc[cpc * i + nb]+2];
      float center_distance = sqrt(comp1 * comp1 + comp2 * comp2 + comp3 * comp3);

      // the overlap distance (how much one penetrates in the other)
      float delta = r1 + r2 - center_distance;

      if (delta < 0) {
        result[3*i + 0] = 0;
        result[3*i + 1] = 0;
        result[3*i + 2] = 0;
        continue;
      }

      // to avoid a division by 0 if the centers are (almost) at the same
      //  location
      if (center_distance < 0.00000001) {
        result[3*i + 0] = 42;
        result[3*i + 1] = 42;
        result[3*i + 2] = 42;
        continue;
      }

      // the force itself
      float r = (r1 * r2) / (r1 + r2);
      float gamma = 1; // attraction coeff
      float k = 2;     // repulsion coeff
      float f = k * delta - gamma * sqrt(r * delta);

      float module = f / center_distance;
      result[3*i + 0] = module * comp1;
      result[3*i + 1] = module * comp2;
      result[3*i + 2] = module * comp3;
    }
  }
}
 
int cuda_collide(int N, int cpc, int T, int diameter, int argc) {
  int* indices;
  float* positions;
  float* force;
  float* diameters;

  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&indices, cpc*N*N*N*sizeof(int));
  hipMallocManaged(&positions, 3*N*N*N*sizeof(float));
  hipMallocManaged(&force, 3*N*N*N*sizeof(float));
  hipMallocManaged(&diameters, N*N*N*sizeof(float));
 
  // initialize
  const float space = 10;
  int i = 0;
  for (size_t x = 0; x < N; x++) {
    float x_pos = x * space;
    for (size_t y = 0; y < N; y++) {
      float y_pos = y * space;
      for (size_t z = 0; z < N; z++) {
        positions[3*i+0] = x_pos;
        positions[3*i+1] = y_pos;
        positions[3*i+2] = z * space;
        i++;
      }
    }
  }

  for (int j = 0; j < N*N*N; j++) {
    diameters[j] = diameter;
  }

  if (argc == 5) {
    // make random accessable pattern
    make_rai(indices, N, cpc);
    std::cout << "Running on GPU (CUDA) for " << T << " iterations (random access pattern)" << std::endl << std::endl;
  } else {
    // make regular accessable pattern
    make_rei(indices, N, cpc);
    std::cout << "Running on GPU (CUDA) for " << T << " iterations (regular access pattern)" << std::endl << std::endl;
  }


  auto t1 = Clock::now();
 
  for (int t = 0; t < T; t++) {
    // Launch kernel on 1M elements on the GPU
    int blockSize = 128;
    int numBlocks = (N*N*N + blockSize - 1) / blockSize;
    collide<<<numBlocks, blockSize>>>(positions, diameters, force, indices, N, cpc);
   
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
  }
 
  auto t2 = Clock::now();
  std::cout << "\033[1mExecution time = "
            << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1)
                   .count()
            << " ms\033[0m" << std::endl;
  std::cout << "Total force = " << compute_sum(force, N) << std::endl;
 
  // Free memory
  hipFree(indices);
  hipFree(positions);
  hipFree(force);
  hipFree(diameters);

  return 0;
}
